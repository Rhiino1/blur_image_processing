#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define CUDA_CHECK_RETURN(value)                                                                     \
    {                                                                                                \
        hipError_t err = value;                                                                     \
        if (err != hipSuccess)                                                                      \
        {                                                                                            \
            printf("Error %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__); \
            exit(1);                                                                                 \
        }                                                                                            \
    }

struct BITMAP_header
{
    char name[2];
    unsigned int size;
    int garbage;
    unsigned int image_offset;
};

struct DIB_header
{
    unsigned int header_size;
    unsigned int width;
    unsigned int height;
    unsigned short int colorplanes;
    unsigned short int bitsperpixel;
    unsigned int compression;
    unsigned int image_size;
    unsigned int temp[4];
};

struct RGB
{
    unsigned char blue;
    unsigned char green;
    unsigned char red;
};

struct Image
{
    int height;
    int width;
    struct RGB **rgb;
};

// Función que permite leer los datos de los pixeles de una imagen .bmp.
// Se usan estructuras creadas como el RGB que nos permite guardar los tres valores
// de color por pixel.
struct Image readImage(FILE *fp, int height, int width, int thread_count)
{
    struct Image pic;
    int i;
    pic.rgb = (struct RGB **)malloc(height * sizeof(void *));
    pic.height = height;
    pic.width = width;

    for (i = height - 1; i >= 0; i--)
    {
        pic.rgb[i] = (struct RGB *)malloc(width * sizeof(struct RGB));
        fread(pic.rgb[i], width, sizeof(struct RGB), fp);
    }

    return pic;
};

// Función que libera la memoria usada al usar la imagen.
void freeImage(struct Image pic)
{
    int i;
    for (i = pic.height - 1; i >= 0; i--)
    {
        free(pic.rgb[i]);
    }
    free(pic.rgb);
}

__global__ void processImage(unsigned char *out, unsigned char *in, int *width, int *height, int *FILTER_SIZE)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int sum = 0;
    int count = 0;

    if (x >= *width || y >= *height)
        return;
    for (int i = -(*FILTER_SIZE); i <= (*FILTER_SIZE); i++)
    {
        if (x < *width && y < *height)
        {
            if ((x + i < 0) || (x + i >= *width) ||
                (y + i < 0) ||
                (y + i >= *height))
                return;
            sum += in[(x + y * *width) + i];
            count++;
        }
    }
    if (count != 0)
    {
        sum /= (count);
        out[x + y * *width] = sum;
    }
    else
    {
        out[x + y * *width] = in[x + y * *width];
    }

    __syncthreads();
}

// Función que realiza el filtro de box blur. Obtenemos un kernel de tamaño nxn
// y recorremos nuestra matriz de pixeles modificando cada uno de los RGB de cada pixel
// siempre multiplicando por el valor del kernel.
void RGBImageToBlur(struct Image pic, unsigned int FILTER_SIZE, int thread_count)
{

    int i, j;
    int size = pic.width * pic.height * sizeof(unsigned char);

    int *d_filter_size;
    int *d_width;
    int *d_height;

    int filter = FILTER_SIZE;

    //Alojando memoria en el device de variables a usar: filter_size, width, height

    CUDA_CHECK_RETURN(hipMalloc(&d_filter_size, sizeof(unsigned int)));
    CUDA_CHECK_RETURN(hipMalloc(&d_width, sizeof(unsigned int)));
    CUDA_CHECK_RETURN(hipMalloc(&d_height, sizeof(unsigned int)));

    //Copiando las variables al device

    CUDA_CHECK_RETURN(hipMemcpy(d_filter_size, &filter, sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_width, &pic.width, sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_height, &pic.height, sizeof(unsigned int), hipMemcpyHostToDevice));

    // Creando canales RGB del host de entrada y salida

    unsigned char *h_r = (unsigned char *)malloc(size);
    unsigned char *h_g = (unsigned char *)malloc(size);
    unsigned char *h_b = (unsigned char *)malloc(size);

    unsigned char *h_r_n = (unsigned char *)malloc(size);
    unsigned char *h_g_n = (unsigned char *)malloc(size);
    unsigned char *h_b_n = (unsigned char *)malloc(size);

    // Rellenando cada vector de cada color
    int count = 0;
    for (i = 0; i < pic.height; i++)
    {
        for (j = 0; j < pic.width; j++)
        {
            h_r[i * pic.width + j] = pic.rgb[i][j].red;
            h_g[i * pic.width + j] = pic.rgb[i][j].green;
            h_b[i * pic.width + j] = pic.rgb[i][j].blue;
            count++;
        }
    }

    //Crear los canales RGB de salida y de entrada del device y alojarles memoria
    unsigned char *d_r_n;
    unsigned char *d_g_n;
    unsigned char *d_b_n;

    CUDA_CHECK_RETURN(hipMalloc(&d_r_n, size));
    CUDA_CHECK_RETURN(hipMalloc(&d_g_n, size));
    CUDA_CHECK_RETURN(hipMalloc(&d_b_n, size));

    unsigned char *d_r;
    unsigned char *d_g;
    unsigned char *d_b;

    CUDA_CHECK_RETURN(hipMalloc(&d_r, size));
    CUDA_CHECK_RETURN(hipMalloc(&d_g, size));
    CUDA_CHECK_RETURN(hipMalloc(&d_b, size));

    //Copiar vectores RGB desde el host al device

    CUDA_CHECK_RETURN(hipMemcpy(d_r, h_r, size, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_g, h_g, size, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice));

    //Calcular los bloques e hilos a usar

    dim3 threadsPerBlock(thread_count, thread_count);
    dim3 blocksPerGrid(ceil((float)pic.width / threadsPerBlock.x), ceil((float)pic.height / threadsPerBlock.y));

    //Invocar el kernel por cada canal

    processImage<<<blocksPerGrid, threadsPerBlock>>>(d_r_n, d_r, d_width, d_height, d_filter_size);
    processImage<<<blocksPerGrid, threadsPerBlock>>>(d_g_n, d_g, d_width, d_height, d_filter_size);
    processImage<<<blocksPerGrid, threadsPerBlock>>>(d_b_n, d_b, d_width, d_height, d_filter_size);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    //Copiar canales RGB desde el device hacia el host

    CUDA_CHECK_RETURN(hipMemcpy(h_r_n, d_r_n, size, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(h_g_n, d_g_n, size, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(h_b_n, d_b_n, size, hipMemcpyDeviceToHost));

    //Pasar los canales RGB hacia la imagen

    for (i = 0; i < pic.height; i++)
    {
        for (j = 0; j < pic.width; j++)
        {
            pic.rgb[i][j].red = h_r_n[i * pic.width + j];
            pic.rgb[i][j].green = h_g_n[i * pic.width + j];
            pic.rgb[i][j].blue = h_b_n[i * pic.width + j];
        }
    }

    //Liberar memoria en device y host

    CUDA_CHECK_RETURN(hipFree(d_filter_size));
    CUDA_CHECK_RETURN(hipFree(d_width));
    CUDA_CHECK_RETURN(hipFree(d_height));

    CUDA_CHECK_RETURN(hipFree(d_r));
    CUDA_CHECK_RETURN(hipFree(d_r_n));

    CUDA_CHECK_RETURN(hipFree(d_g));
    CUDA_CHECK_RETURN(hipFree(d_g_n));

    CUDA_CHECK_RETURN(hipFree(d_b));
    CUDA_CHECK_RETURN(hipFree(d_b_n));

    free(h_r);
    free(h_r_n);

    free(h_g);
    free(h_g_n);

    free(h_b);
    free(h_b_n);
}

//Función que permite crear la imagen de salida y aplicación del filtro.
int createImage(struct BITMAP_header header, struct DIB_header dibheader, struct Image pic, int filter, int kernelSize, char *name, int thread_count)
{
    int i;
    FILE *fpw = fopen(name, "w");
    if (fpw == NULL)
        return 1;

    //Aplicacion del filtro box blur
    if (filter == 2)
    {
        RGBImageToBlur(pic, kernelSize, thread_count);
    }

    fwrite(header.name, 2, 1, fpw);
    fwrite(&header.size, 3 * sizeof(int), 1, fpw);
    fwrite(&dibheader, sizeof(struct DIB_header), 1, fpw); //Puede que sea el problema de archivos corruptos

    for (i = pic.height - 1; i >= 0; i--)
    {
        fwrite(pic.rgb[i], pic.width, sizeof(struct RGB), fpw);
    }
    fclose(fpw);
    return 0;
}

//Función que permite abrir un archivo .bmp y por medio de fread() ir accediendo a los
//bytes del archivo con la información importante para el caso (width, heigth, etc).
void openbmpfile(char *name, char *output, int kernelSize, int thread_count)
{
    FILE *fp = fopen(name, "rb");
    struct BITMAP_header header;
    struct DIB_header dibheader;

    fread(header.name, 2, 1, fp);
    fread(&header.size, 3 * sizeof(int), 1, fp);

    fread(&dibheader, sizeof(struct DIB_header), 1, fp);

    struct Image image = readImage(fp, dibheader.height, dibheader.width, thread_count);

    createImage(header, dibheader, image, 2, kernelSize, output, thread_count);

    fclose(fp);
    freeImage(image);
}

int main(int argc, char *argv[])
{
    struct timeval tval_before, tval_after, tval_result;
    gettimeofday(&tval_before, NULL);

    int thread_count = atoi(argv[3]);

    if (argc != 5)
    {
        printf("Ingrese los valores correctos:\ninput_name output_name threads kernel_size\n");
        return 1;
    }
    openbmpfile(argv[1], argv[2], atoi(argv[4]), thread_count);

    gettimeofday(&tval_after, NULL);

    timersub(&tval_after, &tval_before, &tval_result);
    printf("Time elapsed: %ld.%06ld\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
    printf("done\n");
    return 0;
}